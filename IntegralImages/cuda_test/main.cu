#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include <sys/time.h>

using namespace std;

unsigned long * integralImage(uint8_t*x, int n, int m){
    unsigned long * out = (unsigned long  *)malloc(n*m*sizeof(unsigned long));
    
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < m; ++j)
        {
            unsigned long val = x[i*m + j];
            if (i>=1)
            {
                val += out[(i-1)*m + j];
                if (j>=1)
                {
                    val += out[i*m + j - 1] - out[(i-1)*m + j - 1];
                }
            } else {
                if (j>=1)
                {
                    val += out[i*m + j -1];
                }
            }
            out[i*m + j] = val;
        }
    }
    
    return out;
}

__global__ void image_integral(unsigned long *a, unsigned long *b, int rowsTotal, int colsTotal)
{
    // Thread Ids equal to block Ids because the each blocks contains one thread only.
    int col = blockIdx.x;
    int row = blockIdx.y;
    int temp=0;

    if(col < colsTotal && row < rowsTotal)
    {
        // The first loop iterates from zero to the Y index of the thread which represents the corresponding element of the output/input array.  
        for(int r=0;r<=row;r++)
        {
            // The second loop iterates from zero to the X index of the thread which represents the corresponding element of the output/input array  
            for(int c=0; c<=col; c++)
            {
                temp = temp+a[r*colsTotal+c];
            }
        }
    }

    //Transfer the final result to the output array
    b[row*colsTotal+col]=temp;
}

int main()
{

    int width, height, bpp;
    uint8_t* matrix_a = stbi_load("poppy.jpg", &width, &height, &bpp, 1);
    int total_e = width*height;
    int widthstep = total_e*sizeof(unsigned long);

    unsigned long * a = (unsigned long  *)malloc(widthstep);

    for (int i = 0; i < width *height; ++i)
    {
        a[i] = (unsigned long)matrix_a[i];
    }

    std::cout << "w: " << width << " h: " << height << " b: " << bpp << std::endl;

    std::cout << "Calculating Integral Image..." << std::endl;

    unsigned long * matrix_b= (unsigned long  *)malloc(widthstep);


    for(int r=0;r<height;r++)
    {
        for(int c=0; c<width;c++)
        {
            matrix_b[r*width+c]=0;
        }
    }

    std::cout << "Copied image" << std::endl;

    unsigned long * d_matrix_a, * d_matrix_b;


    hipMalloc(&d_matrix_a,widthstep);
    hipMalloc(&d_matrix_b,widthstep);

    hipMemcpy(d_matrix_a,a,widthstep,hipMemcpyHostToDevice);
    hipMemcpy(d_matrix_b,matrix_b,widthstep,hipMemcpyHostToDevice);

    //Creating a grid where the number of blocks are equal to the number of pixels or input matrix elements.

    //Each block contains only one thread.

    dim3 grid(height,width);

    std::cout << "starting cuda" << std::endl;


    // struct timeval start, end;
    // gettimeofday(&start, NULL);

    image_integral<<<grid,1>>>(d_matrix_a, d_matrix_b,height,width);

    hipDeviceSynchronize();

    // gettimeofday(&end, NULL);

    // double time_tot = ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6;

    // std::cout << "Total time: " << time_tot <<std::endl;

    hipMemcpy(matrix_b,d_matrix_b,widthstep,hipMemcpyDeviceToHost);

    std::cout << "starting serial" << std::endl;

    unsigned long* integral_image = integralImage(matrix_a, height, width);

    std::cout << "finish serial" << std::endl;
int count =0;

    for (int i = 0; i < width*height; ++i)
    {
        if (integral_image[i]!=matrix_b[i])
        {
            //std::cout<<"errore";
            count++;
        }
    }

    std::cout<<count<<std::endl;
    std::cout<<width*height<<std::endl;


    hipFree(d_matrix_a);
    hipFree(d_matrix_b);
    free(a);
    free(matrix_b);

    stbi_image_free(matrix_a);
}