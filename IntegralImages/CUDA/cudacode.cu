#include "hip/hip_runtime.h"
#include "cudacode.h"
#include <hip/hip_runtime.h>

__global__ void image_integral(int *a, int*b, int rowsTotal,int colsTotal)
{
    // Thread Ids equal to block Ids because the each blocks contains one thread only.
    int col = blockIdx.x;
    int row = blockIdx.y;
    int temp=0;

    if(col < colsTotal && row < rowsTotal)
    {
        // The first loop iterates from zero to the Y index of the thread which represents the corresponding element of the output/input array.  
        for(int r=0;r<=row;r++)
        {
            // The second loop iterates from zero to the X index of the thread which represents the corresponding element of the output/input array  
            for(int c=0; c<=col; c++)
            {
                temp = temp+a[r*colsTotal+c];
            }
        }
    }

    //Transfer the final result to the output array
    b[row*colsTotal+col]=temp;
}