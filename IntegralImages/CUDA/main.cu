#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include <sys/time.h>

#include "input_parser.h"

using namespace std;


int print = 0;

void print_help(){
    std::cout << "usage: cuda -p <input image path> [-t <number of threads (int) (default:platform dependent)>] [-json]" << std::endl << std::endl;

    std::cout << "To see this menu again: cuda -h" << std::endl;
}

unsigned long * integralImage(uint8_t*x, int n, int m){
    unsigned long * out = (unsigned long  *)malloc(n*m*sizeof(unsigned long));
    
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < m; ++j)
        {
            unsigned long val = x[i*m + j];
            if (i>=1)
            {
                val += out[(i-1)*m + j];
                if (j>=1)
                {
                    val += out[i*m + j - 1] - out[(i-1)*m + j - 1];
                }
            } else {
                if (j>=1)
                {
                    val += out[i*m + j -1];
                }
            }
            out[i*m + j] = val;
        }
    }
    
    return out;
}

__global__ void sum_rows(unsigned long *a, unsigned long *b, int rowsTotal, int colsTotal, int n_thread)
{
    // Thread Ids equal to block Ids because the each blocks contains one thread only.
    //int col = blockIdx.x;
    int row = blockIdx.x;
    int size_per_thread = rowsTotal/n_thread;
    int start = row*size_per_thread;
    int end = start + size_per_thread;

    if (row==n_thread-1)
    {
        start = (n_thread-1)*size_per_thread;
        end = rowsTotal;
    }
    for (int k = start; k < end; ++k)
    {
            for (int j = 0; j < colsTotal; ++j)
            {
                if (j >=1)
                {
                    b[k*colsTotal + j] = a[k*colsTotal + j] + b[k*colsTotal + j - 1];
                } else {
                    b[k*colsTotal + j] = a[k*colsTotal + j];
                } 
            }
    }

}

__global__ void sum_columns(unsigned long *a, unsigned long *b, int rowsTotal, int colsTotal, int n_thread)
{
    // Thread Ids equal to block Ids because the each blocks contains one thread only.
    int col = blockIdx.x;
    //int row = blockIdx.y;
    int size_per_thread = colsTotal/n_thread;
    int start = col*size_per_thread;
    int end = start + size_per_thread;

    if (col==n_thread-1)
    {
        start = (n_thread-1)*size_per_thread;
        end = colsTotal;
    }
    for (int k = start; k < end; ++k)
    {
        for (int i = 0; i < rowsTotal; ++i)
            {
                if (i >=1)
                {
                    b[i*colsTotal + k] = a[i*colsTotal + k] + b[(i-1)*colsTotal + k];
                } else {
                    b[i*colsTotal + k] = a[i*colsTotal + k];
                } 
        }

    }
}



int main(int argc, char **argv)
{
    InputParser input(argc, argv);
    if(input.cmdOptionExists("-h")){
        print_help();
        return 0;
    }

    if(input.cmdOptionExists("-p")){
        std::string in_file = input.getCmdOption("-p");
        if (in_file == "")
        {
            std::cout << "No input file!\n\n";
            print_help();
            return 2;
        }

        bool json = input.cmdOptionExists("-json");

        int width, height, bpp;
        uint8_t* matrix_a = stbi_load(in_file.c_str(), &width, &height, &bpp, 1);
        int total_e = width*height;
        int widthstep = total_e*sizeof(unsigned long);

        unsigned long * a = (unsigned long  *)malloc(widthstep);

        for (int i = 0; i < width *height; ++i)
        {
            a[i] = (unsigned long)matrix_a[i];
        }

        if (print==1)
        {
            cout << "Input"<<endl;
            for(int r=0;r<height;r++)
            {
                for(int c=0; c<width;c++)
                {
                    cout << a[r*width+c]<<" ";
                }
                cout << endl;
            }
        }

        if(!json){
        std::cout << "w: " << width << " h: " << height << " b: " << bpp << std::endl;

        std::cout << "Calculating Integral Image..." << std::endl;
        }

        unsigned long * matrix_b= (unsigned long  *)malloc(widthstep);
        unsigned long * matrix_t= (unsigned long  *)malloc(widthstep);


        for(int r=0;r<height;r++)
        {
            for(int c=0; c<width;c++)
            {
                matrix_b[r*width+c]=0;
                matrix_t[r*width+c]=0;
            }
        }

        if(!json){
        std::cout << "Copied image" << std::endl;
        }

        unsigned long * d_matrix_a, * d_matrix_b, * d_matrix_t;


        hipMalloc(&d_matrix_a,widthstep);
        hipMalloc(&d_matrix_b,widthstep);
        hipMalloc(&d_matrix_t,widthstep);


        hipMemcpy(d_matrix_a,a,widthstep,hipMemcpyHostToDevice);
        hipMemcpy(d_matrix_b,matrix_b,widthstep,hipMemcpyHostToDevice);
        hipMemcpy(d_matrix_t,matrix_t,widthstep,hipMemcpyHostToDevice);

        if(!json){
        std::cout << "starting cuda" << std::endl;
        }

        struct timeval start, end;
        gettimeofday(&start, NULL);

        int num_thread = 3000;

        
        sum_rows<<<num_thread,1>>>(d_matrix_a, d_matrix_t,height,width, num_thread);
        sum_columns<<<num_thread,1>>>(d_matrix_t, d_matrix_b,height,width, num_thread);

        hipDeviceSynchronize();


        gettimeofday(&end, NULL);

        double time_tot = ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6;

        if(!json){
        std::cout << "Total parallel time: " << time_tot <<std::endl;
        }

        hipMemcpy(matrix_b,d_matrix_b,widthstep,hipMemcpyDeviceToHost);
        hipMemcpy(matrix_t,d_matrix_t,widthstep,hipMemcpyDeviceToHost);


        if (print==1)
        {
            cout << "Cuda Output"<<endl;
            for(int r=0;r<height;r++)
            {
                for(int c=0; c<width;c++)
                {
                    cout << matrix_b[r*width+c]<<" ";
                }
                cout << endl;
            }
        }
        
        if(!json){
        std::cout << "starting serial" << std::endl;
        }

        gettimeofday(&start, NULL);

        unsigned long* integral_image = integralImage(matrix_a, height, width);

        gettimeofday(&end, NULL);

        double time_tot_serial = ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6;

        if(!json){
        std::cout << "Total serial time: " << time_tot_serial <<std::endl;

        std::cout << "finish serial" << std::endl;
        }

        int count =0;

        for (int i = 0; i < width*height; ++i)
        {
            if (integral_image[i]!=matrix_b[i])
            {
                //std::cout<<"errore";
                count++;
            }
        }

        if(!json){
        std::cout<<"Errors ";
        std::cout<<count;
        std::cout<<" over ";
        std::cout<<width*height<<std::endl;
        }

        if (json)
        {
            std::cout << "{time: " << time_tot << ", width: " << width << ", height: " << height << ", errors: " << count << ", time_serial: " << time_tot_serial << "}" << std::endl;   
        }

        hipFree(d_matrix_a);
        hipFree(d_matrix_b);
        free(a);
        free(matrix_b);

        stbi_image_free(matrix_a);
        return 0;
    } else { // no valid arguments
        std::cout << "No input file!\n\n";
        print_help();
        return 1;
    }
}